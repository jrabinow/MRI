#include "hip/hip_runtime.h"
/*
 * C/CUDA implementation of GRASP.
 * Translated from Matlab by Felix Moody, Fall 2014
 * Dependencies:
 *     CUDA compute capability ????, toolkit version ???, driver version ???
 * Input:
 *	Coil sensitivities b1: (image x, image y, coil)
 *	K-space trajectories: (position in k space,
 * Matrices from liver_data.mat (stored in column major format):
 *     b1: 384x384x12 complex doubles
 *     k: 768x600 complex? doubles
 *     kdata: 768x600x12 complex? doubles
 *     w: 768x600 doubles
 * Data requirements:
 *     1st dim b1 = 2nd dim b1
 *     2nd dim k = 2nd dim kdata = 2nd dim w
 *     3rd dim b1 = 3rd dim kdata
 *     1st dim k = 1st dim kdata =  1st dim w = 2 * 1st dim b1
 * So there are 3 variables to data size: nx, ntviews, and nc, and:
 *     b1 = (nx/2, nx/2, nc)
 *     k = (nx, ntviews)
 *     kdata = (nx, ntviews, nc)
 *     w = (nx, ntviews)
 *
 * GRASP Pipeline:
 * Input:
 *	b1: (amplitude, time, coil)
 *	k:
 *	kdata: (amplitutde, time, coil)
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include "hipblas.h" // CUBLAS header
#include <hip/hip_complex.h> // CUDA complex number and operations header

// Macros to convert 2 and 3 dimensional indices to 1 dimensional row major
#define I2D(i,j,j_tot) = (i*j_tot) + j
#define I3D(i,j,k,i_tot,j_tot) = (i*j_tot) + j + (k*i_tot*j_tot)
#define I4D(i,j,k,l,i_tot,j_tot,k_tot) = (i*j_tot) + j + (k*i_tot*j_tot) + (l*i_tot*j_tot*k_tot)

/*
struct param {
    int nite = 8;
    int display = 1;
    double *** kdatau;
    E // MCNUFFT
    y // undersampled data
    W // Total variate dohicky
    lambda
}
*/
/*
static __inline__ void modify(hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-p, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}
*/
/*
void objective(x,dx,t,param) {
    //function res = objective(x,dx,t,param) %**********************************

    // %%%%% L2-norm part
    w=param.E*(x+t*dx)-param.y;
    L2Obj=w(:)'*w(:);

    // %%%%% L1-norm part
    if (param.lambda) {
        w = param.W*(x+t*dx);
        L1Obj = sum((conj(w(:)).*w(:)+param.l1Smooth).^(1/2));
    } else {
        L1Obj=0;
    }

    // %%%%% objective function
    res=L2Obj+param.lambda*L1Obj;
}
*/
/*
void grad(x,param) {
    //function g = grad(x,param)%***********************************************

    // %%%%% L2-norm part
    L2Grad = 2.*(param.E'*(param.E*x-param.y));

    // %%%%% L1-norm part
    if (param.lambda) {
        w = param.W*x;
        L1Grad = param.W'*(w.*(w.*conj(w)+param.l1Smooth).^(-0.5));
    } else {
        L1Grad=0;
    }

    // %%%%% composite gradient
    g=L2Grad+param.lambda*L1Grad;
}
*/
/*
void CSL1NlCg(x0, param) {

    % function x = CSL1NlCg(x0,param)
    %
    % res = CSL1NlCg(param)
    %
    % Compressed sensing reconstruction of undersampled k-space MRI data
    %
    % L1-norm minimization using non linear conjugate gradient iterations
    %
    % Given the acquisition model y = E*x, and the sparsifying transform W,
    % the program finds the x that minimizes the following objective function:
    %
    % f(x) = ||E*x - y||^2 + lambda * ||W*x||_1
    %
    % Based on the paper: Sparse MRI: The application of compressed sensing for rapid MR imaging.
    % Lustig M, Donoho D, Pauly JM. Magn Reson Med. 2007 Dec;58(6):1182-95.
    %
    % Ricardo Otazo, NYU 2008
    %

    printf("\n Non-linear conjugate gradient algorithm");
    printf("\n ---------------------------------------------\n");

    // %%%%% starting point
    ????? x = x0;

    // %%%%% line search parameters
    // WHAT TYPES SHOULD THESE ACTUALLY BE?
    int maxlsiter = 150 ;
    double gradToll = 1e-3 ; // does this work?
    double param.l1Smooth = 1e-15;	// ??
    double alpha = 0.01;
    double beta = 0.6;
    double t0 = 1 ;
    double k = 0;

    // %%%%% compute g0  = grad(f(x))
    g0 = grad(x,param);
    dx = -g0;

    // %%%%% iterations
    while(1) {
        // %%%%% backtracking line-search
	f0 = objective(x,dx,0,param);
	t = t0;
        f1 = objective(x,dx,t,param);
	lsiter = 0;
	while (f1 > f0 - alpha*t*abs(g0(:)'*dx(:)))^2 & (lsiter<maxlsiter) {
		lsiter = lsiter + 1;
		t = t * beta;
		f1 = objective(x,dx,t,param);
	}
	if (lsiter == maxlsiter) {
		disp('Error - line search ...');
		return 1;
	}

	// %%%%% control the number of line searches by adapting the initial step search
	if (lsiter > 2), t0 = t0 * beta;end
	if lsiter<1, t0 = t0 / beta; end

        // %%%%% update x
	x = (x + t*dx);

	// %%%%% print some numbers
        if (param.display) {
            fprintf(' ite = %d, cost = %f \n',k,f1);
        }

        // %%%%% conjugate gradient calculation
	g1 = grad(x,param);
	bk = g1(:)'*g1(:)/(g0(:)'*g0(:)+eps);
	g0 = g1;
	dx =  - g1 + bk* dx;
	k = k + 1;

	// %%%%% stopping criteria (to be improved)
	if (k > param.nite) || (norm(dx(:)) < gradToll), break;end

    }
    return;
}
*/
/*
???? MCNUFFT(k,w,b1) {
    // function  res = MCNUFFT(k,w,b1)
    // k and w here are ku and wu in main, which are the columns of k and w split
    // into nt "frames" of nspokes columns, with frames indexed by the added last dimension
    // so, here k is a 768 x nspokes x nt complex double matrix
    // and w is the same sized double matrix

    % Multicoil NUFFT operator
    % Based on the NUFFT toolbox from Jeff Fessler and the single-coil NUFFT
    % operator from Miki Lustig
    % Input
    % k: k-space trajectory
    % w: density compensation
    % b1: coil sensitivity maps
    %
    % Li Feng & Ricardo Otazo, NYU, 2012

    Nd = [nx,ntviews]; // 3rd dim of b1
    Jd = [6,6];
    Kd = [nx*1.5,ntviews*1.5]
    n_shift = [nx, ntviews]Nd/2; // THIS MEANS 3RD DIM OF B1 MUST BE EVEN
    int tt;
    for (tt=1; tt <= nt; tt++) {
        kk=k(:,:,tt); // take the tt'th frame of k
        om = [real(kk(:)), imag(kk(:))]*2*pi; // separate the real and complex components of the frame and save as 1 dim
        res.st{tt} = nufft_init(om, Nd, Jd, Kd, n_shift,'kaiser'); // run nufft
    }
    res.adjoint = 0;
    res.imSize = size(b1(:,:,1));
    res.dataSize = size(k);
    res.w = sqrt(w);
    res.b1 = b1;
    res = class(res,'MCNUFFT');
}
*/

__global__ void elementWiseMultBySqrt(hipDoubleComplex* kdata, double* w) {
    // Definitely not ideal. Is it bad to only use one thread per block?
    // Also we should only have to compute the squares of the elements of w
    // one time and use the result for all slices of kdata
    int i = blockIdx.x + (blockDim.x * blockIdx.y) + (blockDim.x * blockDim.y * threadIdx.x);
    int j = blockIdx.x + (blockDim.x * blockIdx.y);
    hipDoubleComplex sqrtofelement = make_hipDoubleComplex(sqrt(w[j]), 0);
    // possible overflow error with hipCmul (see hip/hip_complex.h)
    kdata[i] = hipCmul(kdata[i], sqrtofelement); // WARNING
}

int main(int argc,char **argv) {
    int i, j, k, l; // general loop indices
    hipError_t cudaStat; // cuda error type
    hipblasStatus_t stat; // CUBLAS error type
    hipblasHandle_t handle; // handle to CUBLAS context

    // %%%%%% define number of spokes to be used per frame (Fibonacci number)
    int nspokes = 21;

    // %%%%%% load radial data
    // open matrix files and metadata
    FILE *meta_file = fopen("./liver_data/metadata", "rb");
    FILE *b1_file = fopen("./liver_data/b1.matrix", "rb");
    FILE *k_file = fopen("./liver_data/k.matrix", "rb");
    FILE *kdata_file = fopen("./liver_data/kdata.matrix", "rb");
    FILE *w_file = fopen("./liver_data/w.matrix", "rb");

    // load metadata
    size_t dims[3];
    fread(dims, sizeof(size_t), 3, meta_file);

    // %%%%% data dimensions
    int nx = dims[0];
    int ntviews = dims[1];
    int nc = dims[2];

    // set array total lengths
    int b1_total = nx/2 * nx/2 * nc;
    int k_total = nx * ntviews;
    int kdata_total = nx * ntviews * nc;
    int w_total = nx * ntviews;

    // allocate and load b1, k, kdata, and w on CPU
    hipDoubleComplex * b1 = (hipDoubleComplex *)malloc((nx/2)*(nx/2)*nc * sizeof(hipDoubleComplex));
    fread(b1, sizeof(hipDoubleComplex), (nx/2)*(nx/2)*nc, b1_file);
    hipDoubleComplex * k = (hipDoubleComplex *)malloc(nx*ntviews * sizeof(hipDoubleComplex));
    fread(k, sizeof(hipDoubleComplex), nx*ntviews, k_file);
    hipDoubleComplex * kdata = (hipDoubleComplex *)malloc(nx*ntviews*nc * sizeof(hipDoubleComplex));
    fread(kdata, sizeof(hipDoubleComplex), nx*ntviews*nc, kdata_file);
    double * w = (double *)malloc(nx*ntviews * sizeof(double));
    fread(w, sizeof(double), nx*ntviews, w_file);

    // allocate b1, k, kdata, w on GPU
    hipDoubleComplex * b1_d;
    hipDoubleComplex * k_d;
    hipDoubleComplex * kdata_d;
    double * w_d;
    cudaStat = hipMalloc((void**)&b1_d, b1_total*sizeof(*b1));
    cudaStat = hipMalloc((void**)&k_d, kdata_total*sizeof(*k));
    cudaStat = hipMalloc((void**)&kdata_d, kdata_total*sizeof(*kdata));
    cudaStat = hipMalloc((void**)&w_d, w_total*sizeof(*w));
    /*if (cudaStat != hipSuccess |
        cudaStat != cudasuccess |
        cudaStat != hipSuccess |
        cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }*/

    // copy data from CPU to GPU
    cudaStat = hipMemcpy(b1_d, b1, sizeof(b1), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(k_d, k, sizeof(k), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(kdata_d, kdata, sizeof(kdata), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(w_d, w, sizeof(w), hipMemcpyHostToDevice);
    if (cudaStat != hipSuccess) {
        printf("hipMemcpy w failed\n");
        return EXIT_FAILURE;
    } else {
        printf("hipMemcpy w success\n");
    }

    // create cuBLAS context
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    } else {
        printf("CUBLAS initialized\n");
    }

    // b1=b1/max(abs(b1(:)))
    // scale b1 by maximum modulus
    int max_modulus_index;
    stat = hipblasIzamax(handle, b1_total, b1_d, sizeof(hipDoubleComplex), &max_modulus_index);
    const double max_modulus = hipCabs(b1[max_modulus_index]); // hipCabs defined in hip/hip_complex.h
    stat = hipblasZdscal(handle, b1_total, &max_modulus, b1_d, sizeof(hipDoubleComplex));

    // for ch=1:nc,kdata(:,:,ch)=kdata(:,:,ch).*sqrt(w);endc
    // i.e. multiply each of the 12 slices of kdata element wise by sqrt(w)
    dim3 numBlocks(nx, ntviews);
    elementWiseMultBySqrt<<<numBlocks, nc>>>(kdata_d, w_d);


    // %%%%% number of frames
    int nt = floor(ntviews/nspokes) // do we even need floor here?;

    // %%%%% crop the data according to the number of spokes per frame
    // we're basically setting ntviews = nt*nspokes
    // kdata=kdata(:,1:nt*nspokes,:)
    // looping column first due to column major storage
    for (k = 0; k < nc; k++) {
        for (i = 0; i < nx; i++ {
            for (j = 0; j < nt*nspokes; j++) {
                kdata_d[I3D(i,j,k,nx,nt*nspokes)] = kdata_d[I3D(i,j,k,nx,ntviews)];
            }
         }
    }
    // k=k(:,1:nt*nspokes)
    for (i = 0; i < nx; i++ {
        for (j = 0; j < nt*nspokes; j++) {
            k_d[I2D(i,j,nt*nspokes)] = k_d[I2D(i,j,ntviews)];
        }
    }
    // w=w(:,1:nt*nspokes);
    for (i = 0; i < nx; i++ {
        for (j = 0; j < nt*nspokes; j++) {
            w_d[I2D(i,j,nt*nspokes)] = w_d[I2D(i,j,ntviews)];
        }
    }



    // %%%%% sort the data into a time-series
    // divide the 2nd dimension of kdata, k, and w up into nt "frames" of
    // nspokes columns, then index the frames by an added dimension
    // for ii=1:nt
    //     kdatau(:,:,:,ii)=kdata(:,(ii-1)*nspokes+1:ii*nspokes,:);
    //     ku(:,:,ii)=k(:,(ii-1)*nspokes+1:ii*nspokes);
    //     wu(:,:,ii)=w(:,(ii-1)*nspokes+1:ii*nspokes);
    // end
    hipDoubleComplex * kdatau;
    hipDoubleComplex * ku;
    double * w;

    for(l=0;l < nspokes; l++) {
        for(k=0;k < nc;

/*
    // %%%%% multicoil NUFFT operator
    // param.E=MCNUFFT(ku,wu,b1);
    // USE CUFFT FUNCTION HERE

    // %%%%% undersampled data
    // param.y=kdatau;
    // clear kdata kdatau k ku wu w

    // %%%%% nufft recon
    // recon_nufft=param.E'*param.y;

    // %%%%% parameters for reconstruction
    // param.W = TV_Temp();
    // param.lambda = 0.25*max(abs(recon_nufft(:)));
    // param.nite = 8;
    // param.display = 1;

    // fprintf('\n GRASP reconstruction \n')

    // tic
    // recon_cs=recon_nufft;
    // for n=1:3,
    //     recon_cs = CSL1NlCg(recon_cs,param);
    // end
    // toc

    // recon_nufft=flipdim(recon_nufft,1);
    // recon_cs=flipdim(recon_cs,1);

    // %%%%% display 4 frames
    // recon_nufft2=recon_nufft(:,:,1);
    // recon_nufft2=cat(2,recon_nufft2,recon_nufft(:,:,7));
    // recon_nufft2=cat(2,recon_nufft2,recon_nufft(:,:,13));
    // recon_nufft2=cat(2,recon_nufft2,recon_nufft(:,:,23));
    // recon_cs2=recon_cs(:,:,1);
    // recon_cs2=cat(2,recon_cs2,recon_cs(:,:,7));
    // recon_cs2=cat(2,recon_cs2,recon_cs(:,:,13));
    // recon_cs2=cat(2,recon_cs2,recon_cs(:,:,23));

    // figure;
    // subplot(2,1,1),imshow(abs(recon_nufft2),[]);title('Zero-filled FFT')
    // subplot(2,1,2),imshow(abs(recon_cs2),[]);title('GRASP')
*/
/*
    // send matrix to GPU
    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);

    // get matrix from GPU
    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
*/

    // free GPU memory
    hipFree(b1_d);
    hipFree(k_d);
    hipFree(kdata_d);
    hipFree(w_d);

    // destroy cuBLAS context
    //hipblasDestroy(handle);

    // free CPU memory
    free(b1);
    free(k);
    free(kdata);
    free(w);

    // for ch=1:nc,kdata(:,:,ch)=kdata(:,:,ch).*sqrt(w);end
    // this means to multiply each element in each slice of kdata with the
    // square of the corresponding element of w
    //int ch;
    //for(ch = 0; ch < nc; ch++) {
    //    for(
    //    kdata(:,:,ch)=kdata(:,:,ch).*sqrt(w)
    //}
    //printf("%f + %fi\n", creal(b1[3]), cimag(b1[3]));
}
