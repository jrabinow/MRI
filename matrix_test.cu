/*
 * Matrix header test
 * To compile:
 *  $nvcc matrix_test.cu -g -o matrix_test
 *
 * Then to run:
 *  $./matrix_test newHostC
 * or,
 *  $./matrix_test newDeviceC
 * or,
 *  $./matrix_test toDeviceC
 * or,
 *  $./matrix_test crop
 * or,
 *  $./matrix_test cropC
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h> // CUDA complex numbers and operations
#include "matrix.h"
#include "cudaErr.h"


void test_new_matrixC_host() {
	// define dimensions of matrix
	// only need to specify the dimensions we need, up to MAX_DIMS
	size_t dims[MAX_DIMS] = {768, 600};

	// allocate a host matrix with those dimensions
	matrixC * mat = new_matrixC(dims, host); 
	
	// initialize matrix data
	for (size_t i = 0; i < mat->num; i++) {
		mat->data[i] = make_hipDoubleComplex(
				(double)i,
				(double)(i*2));
	}

	// print entries specified by indices
	// keep in mind that matrices are stored
	// in column major format
	print_matrixC(mat, 0, 900);

	// print entries specified by coordinates
	size_t start[MAX_DIMS] = {0, 0};
	size_t end[MAX_DIMS] = {132, 1};
	// C2I takes coordinate array and the
	// matrix dims and converts it to an index
	print_matrixC(mat, C2I(start, mat->dims), C2I(end, mat->dims));
	
	// free matrix
	free_matrixC(mat);
}

void test_new_matrixC_device() {
	// define dimensions of matrix
	// only need to specify the dimensions we need, up to MAX_DIMS
	size_t dims[MAX_DIMS] = {768, 600};

	// allocate a new device matrix with those dimensions
	matrixC * mat = new_matrixC(dims, device); 
	
	// initialize matrix data
	for (size_t i = 0; i < mat->num; i++) {
		mat->data[i] = make_hipDoubleComplex((double)i, (double)(i*2));
	}

	// print entries specified by indices
	// keep in mind that matrices are stored
	// in column major format
	print_matrixC(mat, 0, 900);

	// print entries specified by coordinates
	size_t start[MAX_DIMS] = {0, 0};
	size_t end[MAX_DIMS] = {132, 1};
	// C2I takes coordinate array and the
	// matrix dims and converts it to an index
	print_matrixC(mat, C2I(start, mat->dims), C2I(end, mat->dims));
	
	// free matrix
	free_matrixC(mat);
}

void test_toDeviceC() {
	// define dimensions of matrix
	// only need to specify the dimensions we need, up to MAX_DIMS
	size_t dims[MAX_DIMS] = {768, 600};

	// allocate a new device matrix with those dimensions
	matrixC * mat = new_matrixC(dims, host); 
	
	// initialize matrix data
	for (size_t i = 0; i < mat->num; i++) {
		mat->data[i] = make_hipDoubleComplex((double)i, (double)(i*2));
	}

	// copy matrix to device
	// the host version is preserved
	matrixC * mat_d = toDeviceC(mat);
	
	// print entries
	// matrix is first copied to device,
	// so better to use hosts copy
	print_matrixC(mat_d, 0, 900);
	
	// free device matrix
	free_matrixC(mat_d);

	// free host matrix (because it isn't deleted when sent to device)	
	free_matrixC(mat);
}

void test_crop_matrix() {
	// define dimensions of matrix
	// only need to specify the dimensions we need, up to MAX_DIMS
	size_t dims[MAX_DIMS] = {10, 5};

	// allocate a new host matrix with those dimensions
	matrix * mat = new_matrix(dims, host); 
	
	// initialize matrix data
	for (size_t i = 0; i < mat->num; i++) {
		mat->data[i] = 	(double)i;
	}

	// print matrix before crop
	print_matrix(mat, 0, mat->num);

	// crop matrix
	// the old data is automatically freed
	size_t newDims[MAX_DIMS] = {5, 3};	
	mat = crop_matrix(mat, newDims);
	
	// print entries specified by indices
	// keep in mind that matrices are stored
	// in column major format
	print_matrix(mat, 0, mat->num);
}


void test_crop_matrixC() {
	// define dimensions of matrix
	// only need to specify the dimensions we need, up to MAX_DIMS
	size_t dims[MAX_DIMS] = {10, 5};

	// allocate a new host matrix with those dimensions
	matrixC * mat = new_matrixC(dims, host); 
	
	// initialize matrix data
	for (size_t i = 0; i < mat->num; i++) {
		mat->data[i] = make_hipDoubleComplex(
				(double)i,
				(double)(i*2));
	}

	// print matrix before crop
	print_matrixC(mat, 0, 50);

	// crop matrix
	// the old data is automatically freed
	size_t newDims[MAX_DIMS] = {5, 3};	
	mat = crop_matrixC(mat, newDims);
	
	// print entries specified by indices
	// keep in mind that matrices are stored
	// in column major format
	print_matrixC(mat, 0, 14);
}


int main(int argc, char **argv) {
	if(argc != 2) {
		fprintf(stderr, "Usage: %s ARG\n", argv[0]);
		exit(1);
	}
	if (strcmp(argv[1], "newHostC") == 0) {
		test_new_matrixC_host();
	} else if (strcmp(argv[1], "newDeviceC") == 0) {
		test_new_matrixC_device();
	} else if (strcmp(argv[1], "toDeviceC") == 0) {
		test_toDeviceC();	
	} else if (strcmp(argv[1], "crop") == 0) {
		test_crop_matrix();
	} else if (strcmp(argv[1], "cropC") == 0) {
		test_crop_matrixC();
	} else {
		printf("Not a valid command line argument\n");
	}
}

