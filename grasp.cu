 /*
 * C/CUDA implementation of GRASP.
 * Translated from Matlab by Felix Moody and Julien Rabinow, Fall 2014-Spring 2015
 * Dependencies:
 *     CUDA compute capability ????, toolkit version ???, driver version ???
 *     Developed on Tesla T10 (see "CIMS cuda3 deviceQuery output.txt")
 * To compile with nvcc and cublas
 *     $nvcc grasp.cu -o grasp -lcublas
 * Input: from liver_data.mat (stored in column major format):
 *     Coil sensitivities b1 -- 384x384x12 complex doubles (image x, image y, coil)
 *     K-space trajectories k -- 768x600 complex doubles (position in k space, experiment)
 *     Sample density compensation w: 768x600 doubles (real number between 0 and 1, experiment)
 *     Experimental data kdata: 768x600x12 complex doubles (k space reading, experiment, coil)
 * Data requirements:
 *     1st dim b1 = 2nd dim b1
 *     2nd dim k = 2nd dim kdata = 2nd dim w
 *     3rd dim b1 = 3rd dim kdata
 *     1st dim k = 1st dim kdata =  1st dim w = 2 * 1st dim b1
 * So there are 3 variables to data size: nx, ntviews, and nc, and:
 *     b1 = (nx/2, nx/2, nc)
 *     k = (nx, ntviews)
 *     kdata = (nx, ntviews, nc)
 *     w = (nx, ntviews)
 * Todo/Questions (also caps means questions):
 *     Could we and should we use CUSPARSE instead of cuBLAS?
 *     Can multiple threads access the same memory? At the same time?
 *     How do we think about blocks vs grids? When is it best to break into
 *        blocks and when grids if data fits both?
 *     Should param be global or passed to each subfunction?
 *     Should I break subfunctions into separate files? Why/Why not? How?
 *     Is it worth the space/time to have matrix structs and constructors?
 *         Definitely makes sense to have cudaMalloc packaged into function
 *         Maybe make it an inline function?
 *     In how much generality should we code? For example, should we assume
 *         a certain data size for optimization, or data type?
 *     How to handle errors?
 *     Must the program be designed with a specific data size in mind?
 *     Can we automatically optimize GPU given any data size (in some range)
 *     Not sure if I pulled data from liver_data.mat correctly
 *     In cublas, is it better to take the dot product of a vector with
 *     itself, or to take the norm and then square it?
 * Notes
 *     Any strange complex number stuff is defined in cuComplex.h
 *     Unless otherwise stated, all matlab variables are doubles
 *     %%%%%% = comments from matlab
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h> // for square root
#include <time.h>
#include <complex.h> // C complex numbers and operations; DO I EVEN USE THIS?
#include <hip/hip_runtime.h>
#include "hipblas.h" // CUBLAS
#include <hip/hip_complex.h> // CUDA complex numbers and operations

// Macros to convert multidimensional indices to 1 dimensional row major index
#define I2D(i,j,j_tot) ((i*j_tot) + j)
#define I3D(i,j,k,i_tot,j_tot) ((i*j_tot) + j + (k*i_tot*j_tot))
#define I4D(i,j,k,l,i_tot,j_tot,k_tot) ((i*j_tot) + j + (k*i_tot*j_tot) + (l*i_tot*j_tot*k_tot))

// cuda and cublas error handlers wrappers
#define cudaErrChk(err) { cudaErrorHandler(err, __FILE__, __LINE__); }
#define cublasErrChk(err) {cublasErrorHandler(err, __FILE__, __LINE__); }

// cuda and cublas error handlers
inline void cudaErrorHandler(hipError_t err, const char * file, int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error in %s on line %d:%s\n", file, line, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
inline void cublasErrorHandler(hipblasStatus_t err, const char * file, int line) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS Error in %s on line $d:%s\n", file, line, (const char *)err);
        exit(EXIT_FAILURE);
    }
}

// matrix types
struct mat2D {
    double * d; // the actual data
    int x; // 1st dim size
    int y; // 2nd dim size
    int t; // total # of entries
    int s; // size in bytes of each entry
};

struct mat2DC {
    hipDoubleComplex * d;
    int x;
    int y;
    int t;
    int s;
};

struct mat3D {
    double * d;
    int x; // 1st dim size
    int y; // 2nd dim size
    int z; // 3rd dim size
    int t;
    int s;
};

struct mat3DC {
    hipDoubleComplex * d;
    int x;
    int y;
    int z;
    int t;
    int s;
};

struct mat4D {
    double * d;
    int x; // 1st dim size
    int y; // 2nd dim size
    int z; // 3rd dim size
    int w; // 4th dim size
    int t;
    int s;
};

struct mat4DC {
    hipDoubleComplex * d;
    int x;
    int y;
    int z;
    int w;
    int t;
    int s;
};

// "constructors" for matrix types
mat2D new_mat2D(int xsize, int ysize) {
    mat2D thismat;
    thismat.x = xsize;
    thismat.y = ysize;
    thismat.t = xsize*ysize;
    thismat.s = sizeof(double);
    cudaErrChk(hipMalloc((void**)&(thismat.d), (thismat.t)*(thismat.s)));
    return thismat;
}

mat2DC new_mat2DC(int xsize, int ysize) {
    mat2DC thismat;
    thismat.x = xsize;
    thismat.y = ysize;
    thismat.t = xsize*ysize;
    thismat.s = sizeof(hipDoubleComplex);
    cudaErrChk(hipMalloc((void**)&(thismat.d), (thismat.t)*(thismat.s)));
    return thismat;
}

mat3D new_mat3D(int xsize, int ysize, int zsize) {
    mat3D thismat;
    thismat.x = xsize;
    thismat.y = ysize;
    thismat.z = zsize;
    thismat.t = xsize*ysize*zsize;
    thismat.s = sizeof(double);
    cudaErrChk(hipMalloc((void**)&(thismat.d), (thismat.t)*(thismat.s)));
    return thismat;
}

mat3DC new_mat3DC(int xsize, int ysize, int zsize) {
    mat3DC thismat;
    thismat.x = xsize;
    thismat.y = ysize;
    thismat.z = zsize;
    thismat.t = xsize*ysize*zsize;
    thismat.s = sizeof(hipDoubleComplex);
    cudaErrChk(hipMalloc((void**)&(thismat.d), (thismat.t)*(thismat.s)));
    return thismat;
}

mat4D new_mat4D(int xsize, int ysize, int zsize, int wsize) {
    mat4D thismat;
    thismat.x = xsize;
    thismat.y = ysize;
    thismat.z = zsize;
    thismat.w = wsize;
    thismat.t = xsize*ysize*zsize*wsize;
    thismat.s = sizeof(double);
    cudaErrChk(hipMalloc((void**)&(thismat.d), (thismat.t)*(thismat.s)));
    return thismat;
}

mat4DC new_mat4DC(int xsize, int ysize, int zsize, int wsize) {
    mat4DC thismat;
    thismat.x = xsize;
    thismat.y = ysize;
    thismat.z = zsize;
    thismat.w = wsize;
    thismat.t = xsize*ysize*zsize*wsize;
    thismat.s = sizeof(hipDoubleComplex);
    cudaErrChk(hipMalloc((void**)&(thismat.d), (thismat.t)*(thismat.s)));
    return thismat;
}

// matrix duplicate functions
mat3D copy_mat3D(mat3D in) {
    mat3D thismat = new_mat3D(in.x, in.y, in.z);
    cudaErrChk(hipMemcpy(thismat.d, in.d, in.t*in.s, hipMemcpyDeviceToDevice));
    return thismat;
}

mat3DC copy_mat3DC(mat3DC in) {
    mat3DC thismat = new_mat3DC(in.x, in.y, in.z);
    cudaErrChk(hipMemcpy(thismat.d, in.d, in.t*in.s, hipMemcpyDeviceToDevice));
    return thismat;
}

// matrix print functions
void print2Dmatrix(void *matrix, int dim, int iscomplex, int srow, int scol, int frow, int fcol)
{
	double real_part;
	double imag_part;
	int i, j;
	mat2D *matR;
	mat2DC *matC;

	if(iscomplex) {
		matC = (mat2DC*) matrix;
		for(i = srow; i < frow; i++)
			for(j = scol; j < fcol; j++) {
				real_part = hipCreal(matC->d[I2D(i, j, matC->y)]);
				imag_part = hipCimag(matC->d[I2D(i, j, matC->y)]);
				printf("%f + i*%f ", real_part, imag_part);
			}
				
	} else {
		matR = (mat2D*) matrix;
		for(i = srow; i < frow; i++)
			for(j = scol; j < fcol; j++)
				printf("%f ", matR->d[I2D(i, j, matR->y)]);
	}
}

/*
void print3Dmatrix(void *matrix, int dim, int iscomplex, int srow, int scol, int sslice, int frow, int fcol, int fslice)
{
	double real_part;
	double imag_part;
	int i, j;
	mat2D *matR;
	mat2DC *matC;

	if(iscomplex) {
		matC = (mat2DC*) matrix;
		for(i = tlrow; i < brrow; i++)
			for(j = tlcol; j < brcol; j++) {
				real_part = cuCreal(matC->d[I2D(i, j, matC->y)]);
				imag_part = cuCimag(matC->d[I2D(i, j, matC->y)]);
				printf("%f + i*%f ", real_part, imag_part);
			}
				
	} else {
		matR = (mat2D*) matrix;
		for(i = tlrow; i < brrow; i++)
			for(j = tlcol; j < brcol; j++)
				printf("%f ", matR->d[I2D(i, j, matR->y)]);
	}
}
*/

void printcol_mat3DC(mat3DC mat, int col, int slice) {
    int i;
    hipDoubleComplex elem;
    hipDoubleComplex * mat_cpu = (hipDoubleComplex *)malloc(mat.s*mat.t);
    cudaErrChk(hipMemcpy(mat_cpu, mat.d, mat.s*mat.t, hipMemcpyDeviceToHost));
    for(i = 0; i < mat.x; i++) {
        elem = mat_cpu[I3D(i, col, slice, mat.x, mat.y)];
        printf("%f + %fi\n", hipCreal(elem), hipCimag(elem));
    }
}

    
/*
struct paraxm_type { // ARE THESE THE RIGHT TYPES?
    cuDoubleComplex * E; // MCNUFFT
    cuDoubleComplex * y; // kdatau
    cuDoubleComplex * W; // Total variate dohicky
    double lambda; // trade off control (BETWEEN WHAT?)
    double l1Smooth; // WHAT DOES THIS DO?
    int nite = 8; //
} param;
*/
/*
static __inline__ void modify(cublasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    cublasSscal (handle, n-p, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    cublasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}
*/

/*
// in is a 3 dimensional array on device such that one slice fits in a grid
// and there are fewer slices than threads per block
// y is a 3 dim output array on device of the same size as x
// adjoint is a boolean
__global__ void TV_Temp(cuDoubleComplex * x, cuDoubleComplex * y, int adjoint) {
    if (adjoint == 1) {
        if (blockIdx.x == 1) {
            y[I3D(threadIdx.x, threadIdx.y, 1, blockDim.x, blockDim.y)]
                = -x[I3D(threadIdx.x, threadIdx.y, 1, blockDim.x, blockDim.y)];
        } else if (blockIdx.x == gridDim.x) {
            y[I3D(threadIdx.x, threadIdx.y, gridDim.x, blockDim.x, blockDim.y)]
                = x[I3D(threadIdx.x, threadIdx.y, gridDim.x-1, blockDim.x, blockDim.y)];
        } else {
            y[I3D(threadIdx.x, threadIdx.y, blockId.x, blockDim.x, blockDim.y)]
                = x[I3D(threadIdx.x, threadIdx.y, blockId.x-1, blockDim.x, blockDim.y)]
                - x[I3D(threadIdx.x, threadIdx.y, blockId.x, blockDim.x, blockDim.y)];
        }
    if (adjoint == 0) {
            y[I3D(threadIdx.x, threadIdx.y, blockId.x, blockDim.x, blockDim.y)]
                = x[I3D(threadIdx.x, threadIdx.y, blockId.x+1, blockDim.x, blockDim.y)]
                - x[I3D(threadIdx.x, threadIdx.y, blockId.x, blockDim.x, blockDim.y)];
    }
}
*/
/*
__global__ void L1HelperKernel(cuDoubleComplex * in, double * out, double l1Smooth) {
    // compute index based on block/grid size
    int i =
    out.d[i] = sqrt(cuCabs(in.d[i]) + l1Smooth);
}

// x and dx are 384x384x28 complex double matrices
double objective(cuDoubleComplex * x, cuDoubleComplex * dx, double t) {
    //function res = objective(x,dx,t,param) %**********************************

    // %%%%% L2-norm part
    // w = param.E*(x+t*dx)-param.y;
    // L2Obj=w(:)'*w(:)

    // cast scalars for cuBLAS compatibility
    cuDoubleComplex t_complex = make_cuDoubleComplex(t,(double)0);
    cuDoubleComplex minus1 = make_cuDoubleComplex((double)-1,(double)0);
    // copy x so it doesn't get overwritten
    mat3DC next_x copy_mat3DC(x);
    // next_x=x+t*dx
    cublasZaxpy(handle, x.t, &t_complex, dx.d, dx.s, next_x.d, next_x.s);
    // INSERT FFT HERE
    // mat3DC ft = MCNUFFT(next_x);
    //  ft = ft + (-1)*param.y
    cublasZaxpy(handle, x.t, &minus1, param.y.d, param.y.s, ft.d, ft.s);
    // L2Obj = ft complex dot product ft
    cuDoubleComplex L2Obj;
    cublasZdotc(handle, ft.t, ft.s, ft.t, ft.s, &L2Obj); // IS THIS RIGHT?

    // %%%%% L1-norm part
    // w = param.W*(x+t*dx);
    // L1Obj = sum((conj(w(:)).*w(:)+param.l1Smooth).^(1/2));
    // In matlab code L1Obj wasn't calculated if lambda=0
    mat3DC w = new_mat3DC(next_x.x, next_x.y, next_x.z);
    TV_temp(next_x.d, w.d, 0);
    mat3DC temp = new_mat3D(w.x, w.y, w.z);
    dim3 numBlocks(w.x, w.y);
    L1HelperKernel<<numBlocks, w.z>>(w, temp, param.l1Smooth);
    double L1Obj;
    cublasDasum(handle, temp.t, temp.d, temp.s, &L1Obj);

    // %%%%% objective function
    return L2Obj+param.lambda*L1Obj;
}
*/
/*
mat3DC grad(mat3DC x) {
    // L2-norm part
    // L2Grad =
    // ALLOCATE HERE
    cuDoubleComplex * L2Grad = 2.*(param.E'*(param.E*x-param.y));

    // %%%%% L1-norm part
    if(param.lambda) { // DOES THIS WORK WITH FLOATS?
        // ALLOCATE HERE
        cuDoubleComplex w = param.W*x;
        // v RIGHT TYPE? ALLOCATE
        cuDoubleComplex L1Grad = param.W'*(w.*(w.*conj(w)+param.l1Smooth).^(-0.5));
    } else { // no need to calculate L1Grad if 0 lambda value nullifies it
        return L2Grad;
    }

    //SCALE L1Grad BY LAMBDA WITH CUBLAS FUNCTION

    // %%%%% composite gradient
    return L2Grad+param.lambda*L1Grad;
}
*/

/*
// x0 is a .
mat3DC CSL1NlCg(mat3DC x0, param_type param) {

//  % function x = CSL1NlCg(x0,param)
//  %
//  % res = CSL1NlCg(param)
//  %
//  % Compressed sensing reconstruction of undersampled k-space MRI data
//  %
//  % L1-norm minimization using non linear conjugate gradient iterations
//  %
//  % Given the acquisition model y = E*x, and the sparsifying transform W,
//  % the program finds the x that minimizes the following objective function:
//  %
//  % f(x) = ||E*x - y||^2 + lambda * ||W*x||_1
//  %
//  % Based on the paper: Sparse MRI: The application of compressed sensing for rapid MR imaging.
//  % Lustig M, Donoho D, Pauly JM. Magn Reson Med. 2007 Dec;58(6):1182-95.
//  %
//  % Ricardo Otazo, NYU 2008
//  %

    printf("\n Non-linear conjugate gradient algorithm");
    printf("\n ---------------------------------------------\n");

    // %%%%% starting point
    mat3DC x = copy_mat3DC(x0); // SHOULD I MAKE A COPY OR IS REFERENCE OKAY?

    // %%%%% line search parameters
    int maxlsiter = 150;
    double gradToll = 1e-3;
    param.l1Smooth = 1e-15;
    double alpha = 0.01;
    double beta = 0.6;
    double t0 = 1;
    int k = 0; // iteration counter

    // compute g0  = grad(f(x))
    mat3DC g0 = grad(x);
    mat3DC dx = copy_mat3DC(g0);
    double neg1 = -1.0;
    cublasZdscal(handle, dx.t, &neg1, dx.d, dx.s);


    // %%%%% iterations
    while(1) {
        // %%%%% backtracking line-search
	double f0 = objective(x,dx,0);
	double t = t0;
        double f1 = objective(x,dx,t);
	double lsiter = 0;
        cuDoubleComplex g0dxdotprod;
	while (1) {
                cublasZdotc(handle, g0.t, g0.d, g0.s, dx.d, dx.s, &dotprod);
                if (!(f1 > f0 - alpha*t*cuCabs(dotprod)) || !(lsiter < maxlsiter)) {
                    break;
                }
		lsiter = lsiter + 1.0;
		t = t*beta;
		f1 = objective(x,dx,t);
	}
	if (lsiter == maxlsiter) {
		disp('Error - line search ...');
		return 1;
	}

	// %%%%% control the number of line searches by adapting the initial step search
	if (lsiter > 2) { t0 = t0 * beta; }
	if (lsiter < 1) { t0 = t0 / beta; }

        // %%%%% update x
	// x = (x + t*dx);
        cublasZaxpy(handle, x.t, &make_cuDoubleComplex(t, 0), dx.d, dx.s, x.d, x.s);


	// %%%%% print some numbers
        fprintf("ite = %d, cost = %f\n",k,f1);

        // %%%%% conjugate gradient calculation
	mat3DC g1 = grad(x);
        cuDoubleComplex g1dotprod;
        cuDoubleComplex g0dotprod;
        cublasZdotc(handle, g1.t, g1.d, g1.s, g1.d, g1.s, &g1dotprod);
        cublasZdotc(handle, g0.t, g0.d, g0.s, g0.d, g0.s, &g0dotprod);
        double g1dotprodreal = cuCreal(g1dotprod);
        double g0dotprodreal = cuCreal(g0dotprod);
	double bk = g1dotprodreal/(g0dotprodreal + DBL_EPSILON);
	g0 = g1;
	// dx =  -g1 + bk*dx;
        cublasZdscal(handle, dx.t, &make_cuDoubleComplex(bk, 0.0), dx.d, dx.s);
        cublasZaxpy(handle, g1.t, &neg1,`g1.d, g1.s, dx.d, dx.s);
	k++;

	// %%%%% stopping criteria (to be improved)
        double normdx;
        cublasDznrm2(handle, dx.t, dx.d, dx.s, &normdx);
	if (k > param.nite) || (normdx < gradToll) { break; }
    }
    return x;
}
*/
/*
???? MCNUFFT(k,w,b1) {
    // function  res = MCNUFFT(k,w,b1)
    // k and w here are ku and wu in main, which are the columns of k and w split
    // into nt "frames" of nspokes columns, with frames indexed by the added last dimension
    // so, here k is a 768 x nspokes x nt complex double matrix
    // and w is the same sized double matrix

    % Multicoil NUFFT operator
    % Based on the NUFFT toolbox from Jeff Fessler and the single-coil NUFFT
    % operator from Miki Lustig
    % Input
    % k: k-space trajectory
    % w: density compensation
    % b1: coil sensitivity maps
    %
    % Li Feng & Ricardo Otazo, NYU, 2012

    Nd = [nx,ntviews]; // 3rd dim of b1
    Jd = [6,6];
    Kd = [nx*1.5,ntviews*1.5]
    n_shift = [nx, ntviews]Nd/2; // THIS MEANS 3RD DIM OF B1 MUST BE EVEN
    int tt;
    for (tt=1; tt <= nt; tt++) {
        kk=k(:,:,tt); // take the tt'th frame of k
        om = [real(kk(:)), imag(kk(:))]*2*pi; // separate the real and complex components of the frame and save as 1 dim
        res.st{tt} = nufft_init(om, Nd, Jd, Kd, n_shift,'kaiser'); // run nufft
    }
    res.adjoint = 0;
    res.imSize = size(b1(:,:,1));
    res.dataSize = size(k);
    res.w = sqrt(w);
    res.b1 = b1;
    res = class(res,'MCNUFFT');
}
*/


__global__ void elementWiseMultBySqrt(hipDoubleComplex * kdata, double * w) {
    // We should only have to compute the squares of the elements of w
    // one time and use the result for all slices of kdata
    int i = threadIdx.x * blockIdx.x;
    int j = blockIdx.y;
    hipDoubleComplex sqrtofelement = make_hipDoubleComplex(sqrt(w[i]), 0);
    // possible overflow error with cuCmul (see cuComplex.h)
    kdata[j] = hipCmul(kdata[j], sqrtofelement); // WARNING
}


int main(int argc,char **argv) {
    // Data size (determines gpu optimization, so don't change lightly!)
    int nx = 768;
    int ntviews = 600;
    int nc = 12;

    // GPU block and grid dimensions
    int bt = 512; // max threads per block total
    int bx = 512; // max threads per block x direction
    int by = 512; // max threads per block y direction
    int bz = 64; // max threads per block z direction
    int gx = 65535;
    int gy = 65535;
    int gz = 65535;
    
    int i, j, l, m; // general loop indices (skipped k)

    hipblasHandle_t handle; // handle to CUBLAS context

    cudaErrChk(hipSetDevice(0));

    //  number of spokes to be used per frame (Fibonacci number)
    int nspokes = 21;

    // %%%%%% load radial data
    // open matrix files
    // these were pulled from liver_data.mat by convertmat
    FILE * meta_file = fopen("./liver_data/metadata", "rb");
    FILE * b1_file = fopen("./liver_data/b1.matrix", "rb");
    FILE * k_file = fopen("./liver_data/k.matrix", "rb");
    FILE * kdata_file = fopen("./liver_data/kdata.matrix", "rb");
    FILE * w_file = fopen("./liver_data/w.matrix", "rb");

    // temporarily allocate and load b1, k, kdata, and w on CPU
    hipDoubleComplex * b1_cpu = (hipDoubleComplex *)malloc((nx/2)*(nx/2)*nc * sizeof(hipDoubleComplex));
    fread(b1_cpu, sizeof(hipDoubleComplex), (nx/2)*(nx/2)*nc, b1_file);
    hipDoubleComplex * k_cpu = (hipDoubleComplex *)malloc(nx*ntviews * sizeof(hipDoubleComplex));
    fread(k_cpu, sizeof(hipDoubleComplex), nx*ntviews, k_file);
    hipDoubleComplex * kdata_cpu = (hipDoubleComplex *)malloc(nx*ntviews*nc * sizeof(hipDoubleComplex));
    fread(kdata_cpu, sizeof(hipDoubleComplex), nx*ntviews*nc, kdata_file);
    double * w_cpu = (double *)malloc(nx*ntviews * sizeof(double));
    fread(w_cpu, sizeof(double), nx*ntviews, w_file);

    // allocate b1, k, kdata, w on GPU
    mat3DC b1 = new_mat3DC(nx/2, nx/2, nc);
    mat2DC k = new_mat2DC(nx, ntviews);
    mat3DC kdata = new_mat3DC(nx, ntviews, nc);
    mat2D w = new_mat2D(nx, ntviews);
   
    // copy data from CPU to GPU
    cudaErrChk(hipMemcpy(b1.d, b1_cpu, b1.s*b1.t, hipMemcpyHostToDevice));
    cudaErrChk(hipMemcpy(k.d, k_cpu, k.s*k.t, hipMemcpyHostToDevice));
    cudaErrChk(hipMemcpy(kdata.d, kdata_cpu, kdata.s*kdata.t, hipMemcpyHostToDevice));
    cudaErrChk(hipMemcpy(w.d, w_cpu, w.s*w.t, hipMemcpyHostToDevice));

    // create cuBLAS context
    cublasErrChk(hipblasCreate(&handle));

    // b1=b1/max({|x|: x entry in b1})
    // scale entries of b1 so that maximum modulus = 1
    int max_mod_idx;
    hipDoubleComplex max_mod_num;
    cublasErrChk(hipblasIzamax(handle, b1.t, b1.d, 1, &max_mod_idx));
    cudaErrChk(hipMemcpy(&max_mod_num, &(b1.d[max_mod_idx]), b1.s, hipMemcpyDeviceToHost));
    const double inv_max_mod = 1/hipCabs(max_mod_num);
    cublasErrChk(hipblasZdscal(handle, b1.t, &inv_max_mod, b1.d, 1));

    // WORKING UP TO HERE

/*
    // for ch=1:nc,kdata(:,:,ch)=kdata(:,:,ch).*sqrt(w);endc
    // i.e. multiply each of the 12 slices of kdata element-wise by sqrt(w)
    dim3 numBlocks((kdata.x*kdata.y)/bt, kdata.z);
    elementWiseMultBySqrt<<<numBlocks, bt>>>(kdata.d, w.d);

    printcol_mat3DC(kdata, 0, 0); 

    // %%%%% number of frames
    int nt = ntviews/nspokes; // floor is implicit
*/
/*
    // I THINK THE FOLLOWING SECTION REPLACES THIS
    // %%%%% crop the data according to the number of spokes per frame
    // we're basically setting ntviews = nt*nspokes
    // kdata=kdata(:,1:nt*nspokes,:)
    // looping column first due to column major storage
    for (k = 0; k < nc; k++) {
        for (i = 0; i < nx; i++ {
            for (j = 0; j < nt*nspokes; j++) {
                kdata_d[I3D(i,j,k,nx,nt*nspokes)] = kdata_d[I3D(i,j,k,nx,ntviews)];
            }
         }
    }
    // k=k(:,1:nt*nspokes)
    for (i = 0; i < nx; i++ {
        for (j = 0; j < nt*nspokes; j++) {
            k_d[I2D(i,j,nt*nspokes)] = k_d[I2D(i,j,ntviews)];
        }
    }
    // w=w(:,1:nt*nspokes);
    for (i = 0; i < nx; i++ {
        for (j = 0; j < nt*nspokes; j++) {
            w_d[I2D(i,j,nt*nspokes)] = w_d[I2D(i,j,ntviews)];
        }
    }
*/
/*
    // %%%%% sort the data into a time-series
    // sort kdata, k, and w into time series kdatau, ku, and wu
    // by splitting columns into nt frames of nspokes columns each
    // then index the frames by an added 4th dimension
    // data is cropped in the process (i.e. some columns might not be used)
    // DON'T REMEMBER IF I DID THIS RIGHT
    mat4DC kdatau = new_mat4DC(nx, nspokes, nc, nt);
    mat3DC ku = new_mat3DC(nx, nspokes, nt);
    mat3D wu = new_mat3D(nx, nspokes, nt);
    for (m = 0; m < nt; m++) {
        for (l = 0; l < nc; l++) {
            for (i = 0; i < nx; i++ {
                for (j = 0; j < nspokes; j++) {
                    kdatau.d[I4D(i,j,l,m,nx,nspokes,nc)] = kdata.d[I3D(i,j*m,l,nx,ntviews)];
                }
             }
        }
    }
    for (l = 0; l < nt; l++) {
        for (i = 0; i < nx; i++ {
            for (j = 0; j < nspokes; j++) {
                ku.d[I3D(i,j,l,nx,nspokes)] = k.d[I2D(i,j*l,ntviews)];
            }
        }
    }
    for (l = 0; l < nt; l++) {
        for (i = 0; i < nx; i++ {
            for (j = 0; j < nspokes; j++) {
                wu.d[I3D(i,j,l,nx,nspokes)] = w.d[I2D(i,j*l,ntviews)];
            }
        }
    }
*/
/*
    // %%%%% multicoil NUFFT operator
    param.E=MCNUFFT(ku,wu,b1);
*/
/*
    // %%%%% undersampled data
    param.y=kdatau;
    // clear kdata kdatau k ku wu w
*/
/*
    // %%%%% nufft recon
    // ' := conjugate transpose; * := matrix multiplication
    // ' and * are overloaded, defined in @MCNUFFT
    // what's the order of operations, or does it matter?
    mat3DC recon_nufft=param.E'*param.y;
*/
/*
    // %%%%% parameters for reconstruction
    param_type param;
    // param.W = TV_Temp(); (use TV_Temp kernel)
    //param.lambda = 0.25*max(abs(recon_nufft(:)));
    stat = cublasIZamax(handle, b1_total, b1_d, sizeof(cuDoubleComplex), &max_modulus_index);
    const double max_modulus = cuCabs(b1[max_modulus_index]); // cuCabs defined in cuComplex.h
    param.nite = 8;
    param.display = 1;
*/
    // fprintf('\n GRASP reconstruction \n')

    // long starttime = clock_gettime(CLOCK_MONOTONIC, tv_nsec);
    // mat3DC recon_cs=recon_nufft;
    // for (i = 0; i < 4; i++) {
    //     recon_cs = CSL1NlCg(recon_cs,param);
    // }
    // long elapsedtime = (clock_gettime(CLOCK_MONOTONIC, tv_nsec) - starttime)/1000000;

    // recon_nufft=flipdim(recon_nufft,1);
    // recon_cs=flipdim(recon_cs,1);

    // %%%%% display 4 frames
    // recon_nufft2=recon_nufft(:,:,1);
    // recon_nufft2=cat(2,recon_nufft2,recon_nufft(:,:,7));
    // recon_nufft2=cat(2,recon_nufft2,recon_nufft(:,:,13));
    // recon_nufft2=cat(2,recon_nufft2,recon_nufft(:,:,23));
    // recon_cs2=recon_cs(:,:,1);
    // recon_cs2=cat(2,recon_cs2,recon_cs(:,:,7));
    // recon_cs2=cat(2,recon_cs2,recon_cs(:,:,13));
    // recon_cs2=cat(2,recon_cs2,recon_cs(:,:,23));



    // figure;
    // subplot(2,1,1),imshow(abs(recon_nufft2),[]);title('Zero-filled FFT')
    // subplot(2,1,2),imshow(abs(recon_cs2),[]);title('GRASP')



/*
    // get matrix from GPU
    stat = cublasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != CUBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        cudaFree (devPtrA);
        cublasDestroy(handle);
        return EXIT_FAILURE;
    }
*/

    // free GPU memory
    hipFree(b1.d);
    hipFree(k.d);
    hipFree(kdata.d);
    hipFree(w.d);

    // destroy cuBLAS context
    hipblasDestroy(handle);

    // free CPU memory
    free(b1_cpu);
    free(k_cpu);
    free(kdata_cpu);
    free(w_cpu);
}
